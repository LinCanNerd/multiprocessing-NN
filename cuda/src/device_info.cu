#include <iostream>
#include <hip/hip_runtime.h>

int printDeviceInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << "MB" << std::endl;
        std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "Max Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "Max Grid Size: (" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", " << deviceProp.maxGridSize[2] << ")" << std::endl;
        std::cout << "Max Block Size: (" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << ")" << std::endl;
        std::cout << std::endl;
    }

    return 0;
}
