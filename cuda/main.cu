#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "include/device_info.h"
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "include/data_loading.h"

void checkCudaError(hipError_t error){
    if (error != hipSuccess) {
        printf("Error: %s:%d, ", __FILE__, __LINE__);
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

__global__ void cuda_hello(){
    printf("Block ID: (%d, %d, %d), Thread ID: (%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);


int main() {
    printf("Hello World from CPU!\n");
    printDeviceInfo();
    char* filename = "/home/tintando/Documents/multiprocessing-NN/cuda/datasets/california.csv";
    Sample* samples = readDataset(filename);
    printSamples(samples, 5);


    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}