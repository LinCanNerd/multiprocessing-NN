#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "include/device_info.h"
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "include/data_loading.h"
#include <hiprand/hiprand_kernel.h>



#define N_FEATURES 8
#define N_LABELS 1
#define TILE_SIZE 16


struct MLP {
    int n_layers;
    int batch_size;
    int* layers; //array of hidden layer sizes
    float** weights; //array of pointers to arrays of floats (linearized matrices)
    float** biases; //array of pointers to arrays of floats
    float** activations; //array of pointers to arrays of floats
    float** logits; //array of pointers to arrays of floats
    float** gradients; //array of pointers to arrays of floats (linearizzed matrices)
    float** deltas; //array of pointers to arrays of floats
    float* inputs; //array of floats
    float* labels; //array of floats
};

// in case of a error, it will print:
//  -the file where the error was,
//  -the line that generated the error
//  -the line of code itself
//  -the error
void checkCudaError(hipError_t error){
    if (error != hipSuccess) {
        printf("Error: %s:%d, ", __FILE__, __LINE__);
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}


float** d_allocateWeightShaped(int* layers, int n_layers){
    float** d_weights;// pointer to array of 2d matrix of weights in the device, each matrix[i] = weights between neurons of layer[i] and layer[i-1]
    // allocating memory for weights in the device, store the pointer in d_weights
    // allocates memory for weights of each layer
    checkCudaError(hipMalloc((void**)&d_weights, (n_layers + 1) * sizeof(float*)));
    float** weights = (float**)malloc((n_layers + 1) * sizeof(float*));// pointer to matrix of weights in the device
    for(int i = 0; i < n_layers + 1; i++) {
        // weights[layer]= 2D matrix: entry[i][j]
        int n_rows = i == 0 ? N_FEATURES : layers[i - 1];// neurons of previous layer
        int n_cols = i == n_layers ? N_LABELS : layers[i];// neurons of this layer
        // printf("Weights %d: %d x %d\n", i, n_rows, n_cols);
        checkCudaError(hipMalloc(&weights[i], n_rows * n_cols * sizeof(float)));
        if(weights[i] == NULL) {
            printf("Memory allocation failed.\n");
            return NULL;
        }
    }
    hipMemcpy(d_weights, weights, (n_layers + 1) * sizeof(float*), hipMemcpyHostToDevice);
    return d_weights;
}

float** d_allocateBiasShaped(int* layers, int n_layers){
    float** d_biases;
    checkCudaError(hipMalloc((void**)&d_biases, n_layers * sizeof(float*)));
    float** biases = (float**)malloc(n_layers * sizeof(float*));
    for(int i = 0; i < n_layers; i++) {
        int n_cols = layers[i];
        // printf("Biases %d: %d\n", i, n_cols);
        checkCudaError(hipMalloc(&biases[i], n_cols * sizeof(float)));
    }
    hipMemcpy(d_biases, biases, n_layers * sizeof(float*), hipMemcpyHostToDevice);
    return d_biases;
}

float** d_allocateActivationShaped(int* layers, int n_layers, int batch_size){
    float** d_activations;
    checkCudaError(hipMalloc((void**)&d_activations, n_layers * sizeof(float*)));
    float** activations = (float**)malloc(n_layers * sizeof(float*)); //array of pointers to activation matrices on device
    for(int i = 0; i < n_layers; i++) {
        int n_cols = layers[i];
        // printf("Activations %d: %d x %d\n", i, batch_size, n_cols);
        checkCudaError(hipMalloc(&activations[i], batch_size * n_cols * sizeof(float)));
        hipMemset(activations[i], 0, batch_size * n_cols * sizeof(float));
    }
    hipMemcpy(d_activations, activations, n_layers * sizeof(float*), hipMemcpyHostToDevice);
    return d_activations;
}


__global__ void gpu_print_features(float* features){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx == 4){
        printf("Sample %d: ", idx);
        for(int i = 0; i < 8; i++){
            printf("%f ", features[idx * 8 + i]);
        }
        printf("\n");
    }
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}


__global__ void d_initializeWeights(int layer, float** weights, int n_values, float range, hiprandState *state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n_values){
        hiprandState localState = state[idx];
        float random = hiprand_uniform(&localState);
        // printf("L%d, Thread %d/%d: %f\n", layer, blockIdx.x, idx, random);
        weights[layer][idx] = (random * 2 - 1) * range;
        // weights[layer][idx] = (float) idx;
    }
}

__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, int BRows,
    int BCols, int CRows, int CCols)
{
    float CValue = 0;

    int Row = blockIdx.y*TILE_SIZE + threadIdx.y;
    int Col = blockIdx.x*TILE_SIZE + threadIdx.x;

    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    for (int k = 0; k < (TILE_SIZE + ACols - 1)/TILE_SIZE; k++) {

         if (k*TILE_SIZE + threadIdx.x < ACols && Row < ARows)
             As[threadIdx.y][threadIdx.x] = A[Row*ACols + k*TILE_SIZE + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_SIZE + threadIdx.y < BRows && Col < BCols)
             Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_SIZE + threadIdx.y)*BCols + Col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_SIZE; ++n)
             CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < CRows && Col < CCols)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols) +
           (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
}

__global__ void logits_add_biases_activation_sigmoid(MLP mlp, int i, int n_cols){
    float* logits = mlp.logits[i];
    float* biases = mlp.biases[i];
    float* activations = mlp.activations[i];
    int n_rows = mlp.batch_size;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_values = n_rows * n_cols;
    // int col = idx % n_cols;
    // int row = idx / n_cols;
    if(idx < n_values){
        // printf("adding bias[%d] to logit[%d][%d]\n", col, row, col);
        // double check if modulo or div
        logits[idx] += biases[idx%n_cols]; //adds bias of the row
        // 1 2 3 4
        // 1 2 3 5
        // 1 2 3 6
        // 1 2 3 7

        if (i != mlp.n_layers - 1) {
            //Sigmoid
            activations[idx] = 1 / (1 + exp(-logits[idx]));
        } else {
            activations[idx] = (logits[idx] > 0) ? (logits[idx]) : (0);
        }
    }
}

__global__ void forward_pass(const MLP mlp, int batch_size, int start){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_layers = mlp.n_layers;
    int* layers = mlp.layers;
    float** weights = mlp.weights;
    float** biases = mlp.biases;
    float** activations = mlp.activations;
    float** logits = mlp.logits;
    float* inputs = mlp.inputs;
    float* labels = mlp.labels;
    // printf(" ", idx);
    // Forward pass

    for(int i = 0; i < n_layers; i++){
    // for(int i = 0; i < 1; i++){
        int n_rows = i == 0 ? N_FEATURES : layers[i - 1];
        int n_cols = i == n_layers ? N_LABELS : layers[i];
        
        dim3 blockSize(TILE_SIZE, TILE_SIZE);
        dim3 gridSize((n_cols + blockSize.x - 1) / blockSize.x, (batch_size + blockSize.y - 1) / blockSize.y);
        // Compute logits
        // printf("Computing logits for layer %d\n", i);

                // Print weights[i]
        printf("LAYER %d\n", i);
        printf("weights[%d]: %dx%d\n", i, n_rows, n_cols);
        for(int j = 0; j < n_rows; j++){
            for(int k = 0; k < n_cols; k++){
                printf("%f ", weights[i][j * n_cols + k]);
            }
            printf("\n");
        }
        printf("\n");

        // Print inputs
        printf("inputs: %dx%d\n", batch_size, n_rows);
        for(int j = 0; j < batch_size; j++){
            for(int k = 0; k < n_rows; k++){
                if (i == 0) {
                    printf("%f ", inputs[(start + j) * N_FEATURES + k]);
                } else {
                    printf("%f ", activations[i-1][j * n_rows + k]);
                }
                // printf("%f ", inputs[(start + j) * N_FEATURES + k]);
            }
            printf("\n");
        }
        printf("\n");

        if (i == 0) { // first layer
            MatMul<<<gridSize, blockSize>>>(inputs+start*N_FEATURES*sizeof(float), weights[i], logits[i], batch_size, n_rows, n_rows, n_cols, batch_size, n_cols);
        }
        else {
            MatMul<<<gridSize, blockSize>>>(activations[i-1], weights[i], logits[i], batch_size, n_rows, n_rows, n_cols, batch_size, n_cols);
        }
        hipDeviceSynchronize();
        printf("logits[%d]:\n ", i);
        for(int j = 0; j < batch_size; j++){
            for(int k = 0; k < n_cols; k++){
                printf("%f ", logits[i][j * n_cols + k]);
            }
            printf("\n ");
        }
        printf("\n");
        
        // Add biases to logits and compute activations
        int n_values = batch_size * n_cols;
        int threads_per_block = 256;
        int blocks_per_grid = (n_values + threads_per_block - 1) / threads_per_block;
        logits_add_biases_activation_sigmoid<<<blocks_per_grid, threads_per_block>>>(mlp, i, n_cols);
        hipDeviceSynchronize();
        
    }

}
__global__ void compute_deltas(MLP mlp, int start){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_layers = mlp.n_layers;
    int batch_size = mlp.batch_size;
    int* layers = mlp.layers;
    float** weights = mlp.weights;
    float** activations = mlp.activations;
    float** logits = mlp.logits;
    float** deltas = mlp.deltas;
    float* inputs = mlp.inputs;
    float* labels = mlp.labels + start * N_LABELS * sizeof(float);

    // Compute deltas for the output layer
    int n_rows = batch_size;
    int n_cols = layers[n_layers - 1];
    int n_values = n_rows * n_cols;

    if (idx < n_values){
        int col = idx % n_cols;
        int row = idx / n_cols;
        
        // (predicted-target) [hadamard] step(logits)
        deltas[n_layers - 1][row * n_cols + col] = (activations[n_layers-1][row * n_cols + col] - labels[row * n_cols + col]) * (activations[n_layers - 1][row * n_cols + col] >= 0) ? (1) : (0); 
        // deltas[n_layers - 1][row * n_cols + col] = (float)idx;
        // Print deltas



        // Compute deltas for the hidden layers
        for(int i = n_layers - 2; i > 0; i--){
            // int n_rows = layers[i - 1];
            int n_cols = layers[i];
            int n_values = n_rows * n_cols;
            float sum = 0;
            if (idx < n_values){
                // for(int k = 0; k < layers[i + 1]; k++){
                //     sum += weights[i + 1][idx * layers[i + 1] + k] * deltas[i + 1][k];
                // }
                // if (idx == 0) {
                //     deltas[i][idx] = (float)42069;
                // } else {
                //     deltas[n_layers - 1][row * n_cols + col] = (activations[n_layers-1][row * n_cols + col] - labels[row * n_cols + col]);
                // }
                //d^(l+1)*W^l+1^T [hadamard] afunc'(logit^l)
                //BSIZExLAYERS[i+1] LAYERS[i+1]xLAYERS[i] = BSIZExLAYERS[i] [hadamard] sig(logit^l)(1-sig(logit^l))
                //for every logits compute sig*1-sig, for value in delta multiply by corresponding logit
                //resulting delta contains on each row the deltas for a sample, and columns the neurons
                if (idx == 0){
                    // matmul
                    dim3 blockSize(TILE_SIZE, TILE_SIZE);
                    dim3 gridSize((n_cols + blockSize.x - 1) / blockSize.x, (batch_size + blockSize.y - 1) / blockSize.y);
                    MatMul<<<gridSize, blockSize>>>(deltas[i+1], weights[i+1], deltas[i], batch_size, layers[i+1], layers[i+1], layers[i], batch_size, layers[i]);
                    hipDeviceSynchronize();
                }
                __syncthreads();
                deltas[i][idx] *= (1 - logits[i][idx]) * logits[i][idx];
                // each thread computes sig*1-sig and multiplies to their logit
                // deltas[i][idx] = (1 - activations[i][idx]) * activations[i][idx]
                __syncthreads();
            }
        }
        if (idx == 0) {
            printf("deltas[%d]:\n", idx);
            for(int i = 0; i < n_layers; i++){
                int n_rows = batch_size;
                int n_cols = layers[i];
                for(int j = 0; j < n_rows; j++){
                    for(int k = 0; k < n_cols; k++){
                        printf("%f ", deltas[i][j * n_cols + k]);
                    }
                    printf("\n");
                }
                printf("\n");
            }
            printf("deltas[%d]: %f\n", idx, deltas[n_layers - 1][row * n_cols + col]);
            __syncthreads();
        }
    }
}
__global__ void backpropagation(MLP mlp, int start){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_layers = mlp.n_layers;
    int* layers = mlp.layers;
    float** weights = mlp.weights;
    float** biases = mlp.biases;
    float** activations = mlp.activations;
    float** logits = mlp.logits;
    float** gradients = mlp.gradients;
    float** deltas = mlp.deltas;
    float* inputs = mlp.inputs;
    float* labels = mlp.labels;

    // Deltas
    int n_values = mlp.layers[mlp.n_layers - 2] * mlp.layers[mlp.n_layers - 1];
    int threads_per_block = 256;
    int blocks_per_grid = (n_values + threads_per_block - 1) / threads_per_block;
    compute_deltas<<<blocks_per_grid, threads_per_block>>>(mlp, start);
    // Bias gradients

    // Weight gradients
    // // Backward pass
    // for(int i = n_layers - 1; i >= 0; i--){
    //     int n_rows = i == 0 ? N_FEATURES : layers[i - 1];
    //     int n_cols = i == n_layers ? N_LABELS : layers[i];
    //     int n_values = n_rows * n_cols;
    //     int threads_per_block = 256;
    //     int blocks_per_grid = (n_values + threads_per_block - 1) / threads_per_block;
    //     if(i == n_layers - 1){
    //         // Compute deltas for the output layer
    //         // printf("Computing deltas for output layer\n");
    //         for(int j = 0; j < n_values; j++){
    //             deltas[i][j] = (activations[i][j] - labels[j]) * (1 - activations[i][j]) * activations[i][j];
    //         }
    //         // printf("deltas[%d]:\n ", i);
    //         // for(int j = 0; j < n_rows; j++){
    //         //     for(int k = 0; k < n_cols; k++){
    //         //         printf("%f ", deltas[i][j * n_cols + k]);
    //         //     }
    //         //     printf("\n ");
    //         // }
    //         // printf("\n");
    //     }
    //     else{
    //         // Compute deltas for the hidden layers
    //         // printf("Computing deltas for hidden layer %d\n", i);
    //         int n_cols_next = layers[i + 1];
    //         int n_values_next = n_cols_next * n_cols;
    //         int threads_per_block = 256;
    //         int blocks_per_grid = (n_values + threads_per_block - 1) / threads_per_block;
    //         for(int j = 0; j < n_values; j++){
    //             float sum = 0;
    //             for(int k = 0; k < n_values_next; k++){
    //                 sum += weights[i + 1][k] * deltas[i + 1][k];
    //             }
    //             deltas[i][j] = (1 - activations[i][j]) * activations[i][j] * sum;
        

        
}

void train(const MLP mlp, int epochs, int batch_size, int n_samples){
    printf("Training...\n");

    int n_layers = mlp.n_layers;
    int* h_layers = (int*)malloc(n_layers * sizeof(int));

    hipMemcpy(h_layers, mlp.layers, n_layers * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < n_layers; i++) {
        printf("Layer %d: %d\n", i, h_layers[i]);
    }


    for(int i = 0; i < epochs; i++){     // for(int i = 0; i < 1; i++){

        printf("Epoch %d\n", i);
        // for (int start = 0; start < n_samples - batch_size; start += batch_size) {         // for (int start = 0; start < 1; start += batch_size) {
        for (int start = 0; start < 1; start += batch_size) {
        // Forward pass 
            forward_pass<<<1,1>>>(mlp, batch_size, start);
            hipDeviceSynchronize();

        // Backward pass
            backpropagation<<<1,1>>>(mlp, start);
            hipDeviceSynchronize();
        }
    }
}

void printWeights(const MLP mlp) {
    int n_layers = mlp.n_layers;
    float** h_weights = (float**)malloc(n_layers * sizeof(float*));
    int* h_layers = (int*)malloc(n_layers * sizeof(int));

    hipMemcpy(h_weights, mlp.weights, n_layers * sizeof(float*), hipMemcpyDeviceToHost);
    hipMemcpy(h_layers, mlp.layers, n_layers * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < n_layers; i++) {
        int n_rows = i == 0 ? N_FEATURES : h_layers[i - 1];
        int n_cols = h_layers[i];
        int n_values = n_rows * n_cols;

        float* h_weights_i = (float*)malloc(n_values * sizeof(float));
        if (h_weights_i == NULL) {
            printf("Memory allocation failed.\n");
            return;
        }

        hipMemcpy(h_weights_i, h_weights[i], n_values * sizeof(float), hipMemcpyDeviceToHost);

        printf("Weights for Layer %d: %dx%d\n", i, n_rows, n_cols);
        for (int j = 0; j < n_rows; j++) {
            for (int k = 0; k < n_cols; k++) {
                printf("%.2f ", h_weights_i[j * n_cols + k]);
            }
            printf("\n");
        }
        printf("\n");

        free(h_weights_i);
    }
    free(h_weights);
    free(h_layers);
}

__global__ void print_layers(const MLP mlp){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx == 0){
        printf("Number of layers: %d\n", mlp.n_layers);
        for(int i = 0; i < mlp.n_layers; i++){
            printf("Layer %d: %d\n", i, mlp.layers[i]);
        }
    }
}

__global__ void d_printActivationShaped(MLP mlp){
    for (int i = 0; i < mlp.n_layers; i++) {
        int n_rows = mlp.batch_size;
        int n_cols = mlp.layers[i];
        int n_values = n_rows * n_cols;

        printf("Activations for Layer %d: %dx%d\n", i, n_rows, n_cols);
        for (int j = 0; j < n_rows; j++) {
            for (int k = 0; k < n_cols; k++) {
                printf("%.2f ", mlp.activations[i][j * n_cols + k]);
            }
            printf("\n");
        }
        printf("\n");
    }
}


void printActivationShaped(const MLP mlp, int* layers, int choice) {
    float** h_activations = (float**)malloc(mlp.n_layers * sizeof(float*)); // activation shaped, may contain logits, activations, gradients, deltas
    // int* h_layers = (int*)malloc(mlp.n_layers * sizeof(int));
    // int* h_layers = layers;
    char* parameter;
    switch (choice) {
        case 1: // Copy mlp.logits
            hipMemcpy(h_activations, mlp.logits, mlp.n_layers * sizeof(float*), hipMemcpyDeviceToHost);
            parameter = "Logits";
            break;
        case 2: // Copy mlp.activations
            hipMemcpy(h_activations, mlp.activations, mlp.n_layers * sizeof(float*), hipMemcpyDeviceToHost);
            parameter = "Activations";
            break;
        case 3: // Copy mlp.deltas
            hipMemcpy(h_activations, mlp.deltas, mlp.n_layers * sizeof(float*), hipMemcpyDeviceToHost);
            parameter = "Deltas";
            break;
        case 4: // Copy mlp.gradients
            hipMemcpy(h_activations, mlp.gradients, mlp.n_layers * sizeof(float*), hipMemcpyDeviceToHost);
            parameter = "Gradients";
            break;
        default:
            printf("Invalid choice.\n");
            break;
    }

    // hipMemcpy(h_activations, mlp.activations, mlp.n_layers * sizeof(float*), hipMemcpyDeviceToHost);

    // h_layers = layers;
    for (int i = 0; i < mlp.n_layers; i++) {
        int n_rows = mlp.batch_size;
        int n_cols = layers[i];
        int n_values = n_rows * n_cols;

        float* h_activations_i = (float*)malloc(n_rows * n_cols * sizeof(float));
        if (h_activations_i == NULL) {
            printf("h_activations_i Memory allocation failed.\n");
            exit(1);
        }

        hipMemcpy(h_activations_i, h_activations[i], n_rows * n_cols * sizeof(float), hipMemcpyDeviceToHost);

        printf("%s for Layer %d: %dx%d\n", parameter, i, n_rows, n_cols);
        for (int j = 0; j < n_rows; j++) {
            for (int k = 0; k < n_cols; k++) {
                printf("%.2f ", h_activations_i[j * n_cols + k]);
            }
            printf("\n");
        }
        printf("\n");

        free(h_activations_i);
    }
    free(h_activations);
    // free(h_layers);
}



int main(int argc, char* argv[]) {
    //usage ./main hidden_size1 hidden_size2 ... hidden_sizeN epochs batch_size
   
    printf("Hello World from CPU!\n");
    printDeviceInfo();

    if(argc < 4) {
        printf("Invalid number of arguments. Please provide at least 3 arguments.\n");
        return 1;
    }

    // Allocate memory for the integer array that will store the hidden layer sizes
    
    int n_hidden_layers = argc - 3;// the number of hidden layers
    //array: entry[i] = size of hidden layer i || size of output layer
    int* layers = (int*)malloc((n_hidden_layers+1) * sizeof(int)); 
    if(layers == NULL) {
        printf("Memory allocation failed.\n");
        return 1;
    }

    // Convert arguments to integers and store them in the array
    for(int i = 1; i < argc-2; i++) {
        layers[i - 1] = atoi(argv[i]);
    }
    layers[n_hidden_layers] = N_LABELS; //output layer
    n_hidden_layers++; //TODO: rename for clarity

    int epochs = atoi(argv[argc-2]);
    int batch_size = atoi(argv[argc-1]);

    printf("Hidden layer sizes: ");
    for(int i = 0; i < n_hidden_layers; i++) {
        printf("%d ", layers[i]);
    }
    
    printf("\nEpochs: %d\n", epochs);
    printf("Batch size: %d\n\n", batch_size);


    // Load the dataset (must have 8 features and 1 target)
    char* filename = "/home/tintando/Documents/multiprocessing-NN/cuda/datasets/california.csv";

    // Read the dataset

    int n_samples;// the number of samples
    Sample* samples = readDataset(filename, &n_samples);//array of: (features[8], label)
    printf("Number of samples: %d\n", n_samples);
    printSamples(samples, 5);

    //memory needed 
    //weights 8xhiddensize1 + hiddensize1xhiddensize2 + ... + hiddensize2x1 
    //biases hiddensize1 + hiddensize2 + ... + 1
    //activations batch_size x hiddensize1 + batch_size x hiddensize2 + ... + batch_size x 1
    //deltas batch_size x hiddensize1 + batch_size x hiddensize2 + ... + batch_size x 1
    //outputs batch_size x hiddensize1 + batch_size x hiddensize2 + ... + batch_size x 1
    //inputs batch_size x 8
    //targets batch_size x 1


    MLP mlp;
    // coalesce features and labels
    printf("HOST: Coalescing features and labels...\n");
    
    float* h_features = (float*)malloc(n_samples * N_FEATURES * sizeof(float));//array of features (on the host)
    float* h_labels = (float*)malloc(n_samples * N_LABELS * sizeof(float));//array of labels (on the host)
    for(int i = 0; i < n_samples; i++) {
        //copy sample by sample from host to device
        //can be read as: 
        // move in h_features[i*N_FEATURES] = samples[i].features
        memcpy(h_features + i * N_FEATURES, samples[i].features, N_FEATURES * sizeof(float));
        h_labels[i] = samples[i].label;
    }
    
    printf("HOST: Allocating device memory...\n");
    
    float* d_features;// array of features (on the device)
    float* d_labels;// array of labels (on the device)
    for(int i = 0; i < n_samples; i++) {
    checkCudaError(hipMalloc((void**)&d_features, n_samples * N_FEATURES * sizeof(float))); //allocates features in device memory  
    hipMemcpy(d_features, h_features, n_samples * N_FEATURES * sizeof(float), hipMemcpyHostToDevice);// initializes features in device memory  
    checkCudaError(hipMalloc((void**)&d_labels, n_samples * N_LABELS * sizeof(float)));// allocates labels in device memory  
    hipMemcpy(d_labels, h_labels, n_samples * N_LABELS * sizeof(float), hipMemcpyHostToDevice);// initializes features in device memory  

    // Allocate memory for the weights, biases, activations, logits, gradients, deltas
    printf("HOST: Allocating device memory for weights, biases, activations, logits, gradients, deltas...\n");
    // Allocate memory for the weights
    float** d_weights = d_allocateWeightShaped(layers, n_hidden_layers); //array of pointers to device memory

    // Allocate memory for the biases
    float** d_biases = d_allocateBiasShaped(layers, n_hidden_layers); //array of pointers to device memory

    // Allocate memory for the activations
    float** d_activations = d_allocateActivationShaped(layers, n_hidden_layers, batch_size); //array of pointers to device memory

    // Allocate memory for the logits
    float** d_logits = d_allocateActivationShaped(layers, n_hidden_layers, batch_size); //array of pointers to device memory

    // Allocate memory for the accumulated gradients
    float** d_gradients = d_allocateWeightShaped(layers, n_hidden_layers); //array of pointers to device memory

    // Allocate memory for the deltas
    float** d_deltas = d_allocateBiasShaped(layers, n_hidden_layers); //array of pointers to device memory

    int* d_layers;
    checkCudaError(hipMalloc((void**)&d_layers, n_hidden_layers * sizeof(int)));
    hipMemcpy(d_layers, layers, n_hidden_layers * sizeof(int), hipMemcpyHostToDevice);

    mlp.n_layers = n_hidden_layers;
    mlp.batch_size = batch_size;
    mlp.layers = d_layers;
    mlp.weights = d_weights;
    mlp.biases = d_biases;
    mlp.activations = d_activations;
    mlp.logits = d_logits;
    mlp.gradients = d_gradients;
    mlp.deltas = d_deltas;
    mlp.inputs = d_features;
    mlp.labels = d_labels;

    // int* h_layers = (int*)malloc(mlp.n_layers * sizeof(int));
    // hipMemcpy(h_layers, mlp.layers, mlp.n_layers * sizeof(int), hipMemcpyDeviceToHost);

    
    printf("HOST: Initializing hiprand...\n");
    // Initialize hiprand
    hiprandState* d_state;
    checkCudaError(hipMalloc((void**)&d_state, 1 * sizeof(hiprandState)));


    // Find the two largest layers
    int largest1 = 0;
    int largest2 = 0;
    for (int i = 0; i < n_hidden_layers; i++) {
        if (layers[i] > largest1) {
            largest2 = largest1;
            largest1 = layers[i];
        } else if (layers[i] > largest2) {
            largest2 = layers[i];
        }
    }
    int product = largest1 * largest2;
    

    setup_kernel<<<(product + 31)/32, 32>>>(d_state, 42);
    hipDeviceSynchronize();

    printf("HOST: Initializing weights...\n");
    for (int i = 0; i < n_hidden_layers; i++) {
        int n_rows = i == 0 ? N_FEATURES : layers[i - 1];
        int n_cols = i == n_hidden_layers ? N_LABELS : layers[i];
        float range = 1/sqrt(n_cols);
        int n_values = n_rows * n_cols;

        int block_size = TILE_SIZE*TILE_SIZE;
        int grid_size = (n_values + block_size - 1) / block_size;
        
        // float* d_weights_i = mlp.weights[i]; //pointer to device memory, not HOST!!!!
        d_initializeWeights<<<grid_size, block_size>>>(i, mlp.weights, n_values, range, d_state);
        // hipDeviceSynchronize();
    }


    // Print d_weights
    printWeights(mlp);
    // printActivationShaped(mlp);
    // test_activation<<<1, 1>>>(mlp);
    printActivationShaped(mlp, layers, 1);

    train(mlp, epochs, batch_size, n_samples);

    // d_printActivationShaped<<<1,1>>>(mlp);

    // gpu_print_features<<<1, 32>>>(d_features);

    printActivationShaped(mlp, layers, 1); // logits
    printActivationShaped(mlp, layers, 2); // activations
    // printActivationShaped(mlp, layers, 3); // deltas

    // cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
    }
